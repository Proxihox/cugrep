/**
 * CUDA-accelerated pattern matching implementation of grep
 * Searches for patterns in text files using parallel GPU processing
 */

// Standard library includes

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <queue>
#include <filesystem>
#include <fcntl.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <unistd.h>
#include <chrono>

// Configuration constants
#define MAX_MATCHES 60000    // Maximum number of matches to store
#define CHUNK_SIZE 400       // Size of text chunk per GPU thread
#define debug true          // Enable debug output

namespace fs = std::filesystem;
using namespace std;

// Fast I/O optimization
#define fastio ios_base::sync_with_stdio(false); cin.tie(NULL); cout.tie(NULL);

// Global state
static bool complement = false;   // -v option: invert match
static bool recursive = false;    // -r option: recursive search
static bool case_insensitive = false; // -i option: case insensitive search
static bool match_start = false; //  match start of line
static bool match_end = false;   //  match end of line
static string pattern;           // Pattern to search for
static long long patternSize;    // Cached pattern length
static long long MAX_THREADS;    // Maximum GPU threads to use
static long long h_old;          // Track processed matches count

/**
 * Represents a pattern match location in text
 */
struct Match {
    long long start;    // Start position of match
    long long end;      // End position of match
};

/**
 * GPU configuration structure stored in constant memory
 */
struct GpuConfig {
    char* d_pattern;              // Device pattern string
    long long patternSize;        // Pattern length
    bool complement;              // Invert match flag
    Match *mlist;                // Array to store matches
    unsigned long long *d_counter; // Number of matches found
    bool case_insensitive; // Case insensitive flag
};

// GPU constant memory configuration
__constant__ GpuConfig c_config;

auto timenow(chrono::time_point<chrono::high_resolution_clock> startt, string s = "Time") {
    auto tnow = chrono::high_resolution_clock::now();
    if(debug) {
        fprintf(stderr, "%s : %ld us\n", 
                s.c_str(),
                chrono::duration_cast<chrono::microseconds>(tnow - startt).count());
    }
    return tnow;
}

/**
 * Handles result based on complement flag
 * @param a - Input value to complement if needed
 * @return Original or complemented boolean result
 */
long long result(long long a) {
    return complement ? !bool(a) : a;
}

/**
 * Searches for pattern within line using string comparison
 * @param line - Text line to search in
 * @return Position of match (1-based) or 0 if not found
 */
long long searchline(const string& line) {
    for(long long startPos = 0; startPos < line.size(); startPos++) {
        long long matchLen = 0;
        // Try matching pattern at current position
        while(startPos + matchLen < line.size() && 
              line[startPos + matchLen] == pattern[matchLen]) {
            matchLen++;
            if(matchLen == patternSize) {
                return result(startPos + 1);
            }
        }
    }
    return result(0);
}

/**
 * Processes input stream line by line
 * @param is - Input stream to process
 */
void searchstream(istream& is) {
    string line;
    while(getline(is, line)) {
        if(searchline(line)) {
            cout << line << '\n';
        }
    }
}



/**
 * Processes memory-mapped file data
 * @param data - Pointer to file data
 * @param size - Size of file in bytes
 */

__device__ long long deviceRes(long long a){
    return c_config.complement ? !bool(a) : a;
}

__device__ bool deviceMatch(const char* data, long long startPos){
    long long matchLen = 0;
    if(c_config.case_insensitive){
        while(data[startPos + matchLen] == c_config.d_pattern[matchLen] 
            or data[startPos + matchLen] == c_config.d_pattern[matchLen] - 32) {
            // Check for case-insensitive match
            matchLen++;
            if(matchLen == c_config.patternSize) {
                return deviceRes(startPos + 1);
            }
        }
    }
    else{
        while(data[startPos + matchLen] == c_config.d_pattern[matchLen]) {
            matchLen++;
            if(matchLen == c_config.patternSize) {
                return deviceRes(startPos + 1);
            }
        }
    }
    return deviceRes(0);
}

__device__ bool deviceMatch_CaseInsensitive(const char* data, long long startPos){
    long long matchLen = 0;
    while(data[startPos + matchLen] == c_config.d_pattern[matchLen] 
        or data[startPos + matchLen] == c_config.d_pattern[matchLen] - 32) {
        // Check for case-insensitive match
        matchLen++;
        if(matchLen == c_config.patternSize) {
            return deviceRes(startPos + 1);
        }
    }
    return deviceRes(0);
}


// Multiple Cuda Kernels, optimized for different patterns of search

// Basic string matching
__global__ void cudaSearchFile(const char* data, long long block_size, 
                                long long size_max, long long offset) {
    // Calculate thread's position and chunk
    long long id = threadIdx.x + 1024*blockIdx.x;
    long long iter = id*block_size;
    
    // Find start of current line
    while(iter > 0 && iter < size_max && data[iter] != '\n') iter--;
    
    long long limit = min((block_size)*(id+1), size_max);
    
    // Process lines in thread's chunk
    while(iter < limit) {
        long long startIndex = iter;
        bool matchFound = false;
        
        // Search current line for pattern
        while(iter < limit && iter < size_max && data[iter] != '\n') {
            if(!matchFound && deviceMatch(data, iter)) {
                matchFound = true;
            }
            iter++;
        }
        
        // Store match if found
        if(matchFound && iter < size_max && data[iter] == '\n') {
            long long index = atomicAdd(c_config.d_counter, 1);
            if(index < MAX_MATCHES) {
                c_config.mlist[index] = {offset+startIndex, offset+iter};
            }
        }
        iter++;
    }
}


__global__ void cudaSearchFile_Start(const char* data, long long block_size, 
                                long long size_max, long long offset) {
    // Calculate thread's position and chunk
    long long id = threadIdx.x + 1024*blockIdx.x;
    long long iter = id*block_size;
    
    // Find start of current line
    while(iter > 0 && iter < size_max && data[iter] != '\n') iter--;
    
    long long limit = min((block_size)*(id+1), size_max);
    
    // Process lines in thread's chunk
    while(iter < limit) {
        long long startIndex = iter;
        bool matchFound = false;
        matchFound = deviceMatch(data, startIndex);
        // Search current line for pattern
        while(iter < limit && iter < size_max && data[iter] != '\n') {
            iter++;
        }
        
        // Store match if found
        if(matchFound && iter < size_max && data[iter] == '\n') {
            long long index = atomicAdd(c_config.d_counter, 1);
            if(index < MAX_MATCHES) {
                c_config.mlist[index] = {offset+startIndex, offset+iter};
            }
        }
        iter++;
    }
}

__global__ void cudaSearchFile_End(const char* data, long long block_size, 
                                long long size_max, long long offset) {
    // Calculate thread's position and chunk
    long long id = threadIdx.x + 1024*blockIdx.x;
    long long iter = id*block_size;
    
    // Find start of current line
    while(iter > 0 && iter < size_max && data[iter] != '\n') iter--;
    
    long long limit = min((block_size)*(id+1), size_max);
    
    // Process lines in thread's chunk
    while(iter < limit) {
        long long startIndex = iter;
        bool matchFound = false;
        
        // Search current line for pattern
        while(iter < limit && iter < size_max && data[iter] != '\n') {
            iter++;
        }
        matchFound = deviceMatch(data, iter-c_config.patternSize);
        // Store match if found
        if(matchFound && iter < size_max && data[iter] == '\n') {
            long long index = atomicAdd(c_config.d_counter, 1);
            if(index < MAX_MATCHES) {
                c_config.mlist[index] = {offset+startIndex, offset+iter};
            }
        }
        iter++;
    }
}



void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error at %s: %s\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

/**
 * Opens and memory-maps a file for processing
 * @param filePath - Path to file to process
 */
void fileMap(const string& filePath) {
    // Open file
    if(debug)fprintf(stderr, "File : %s\n", filePath.c_str());
    auto start = chrono::high_resolution_clock::now();
    const char* fname = filePath.c_str();
    long long fd = open(fname, O_RDONLY);
    if (fd == -1) {
        perror("open");
        return;
    }

    // Get file size
    struct stat sb;
    if(fstat(fd, &sb) == -1) {
        perror("fstat");
        close(fd);
        return;
    }

    // Memory map the file into program memory
    char* data = reinterpret_cast<char*>(
        mmap(nullptr, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0)
    );
    if(debug) start = timenow(start,"mmap");
    char* d_data;
    if(data == MAP_FAILED) {
        perror("mmap");
        close(fd);
        return;
    }
    
    // compute total number of threads
    long long total_threads = max(1ll,(long long)(sb.st_size/CHUNK_SIZE + 1));
    long long memiter = 0;
    if(debug) fprintf(stderr,"Total Memory : %lld\n",(long long)sb.st_size);
    if(debug) fprintf(stderr,"Total threads : %lld\n",total_threads);
    hipError_t err = hipMalloc(&d_data, min((long long)MAX_THREADS*CHUNK_SIZE, (long long)sb.st_size));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        munmap(data, sb.st_size);
        close(fd);
        return;
    }

    // Handle all threads, chunk by chunk
    while(total_threads > 0){

        long long n_threads = min(MAX_THREADS,total_threads);
        long long mem_end = min((long long) MAX_THREADS*CHUNK_SIZE + memiter,(long long)sb.st_size);
        long long memsize = min(mem_end - memiter, n_threads*CHUNK_SIZE);
        if(debug)start = timenow(start,"Setup");
        err = hipMemcpy(d_data, data+memiter, memsize, hipMemcpyHostToDevice);
        if(debug)start = timenow(start,"Memcpy");
        if (err != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
            hipFree(d_data);
            munmap(data, sb.st_size);
            close(fd);
            return;
        }

        long long block_size = (sb.st_size+n_threads-1)/n_threads;
        long long n_blocks = (n_threads+1023)/1024;
        long long threads_per_block = min(1024ll,n_threads);
        if(match_start) {
            cudaSearchFile_Start<<<n_blocks,threads_per_block>>>(d_data,block_size,memsize,memiter);
        } else if(match_end) {
            cudaSearchFile_End<<<n_blocks,threads_per_block>>>(d_data,block_size,memsize,memiter);
        } else{
        cudaSearchFile<<<n_blocks,threads_per_block>>>(d_data,block_size,memsize,memiter);
        }
        checkCudaError("kernel launch");
        memiter += memsize;
        total_threads -= n_threads;
        // if(debug) fprintf(stderr, "threads : %lld\n", n_threads);
        // if(debug) fprintf(stderr,"Remaining threads : %lld\n",total_threads);
        // if(debug) fprintf(stderr,"Memsize : %lld\n",memsize);
        hipDeviceSynchronize();
        start = timenow(start,"search");
        checkCudaError("kernel sync");
        if(debug)fprintf(stderr,"\n");
    }

    unsigned long long h_counter;
    Match *h_list = (Match*)malloc(sizeof(Match)*MAX_MATCHES);
    GpuConfig hostConfig;

    hipMemcpyFromSymbol(&hostConfig, HIP_SYMBOL(c_config), sizeof(GpuConfig));
    hipMemcpy(&h_counter, hostConfig.d_counter, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    // Get matches from device
    hipMemcpy(h_list, hostConfig.mlist, sizeof(Match)*h_counter, hipMemcpyDeviceToHost);
    
    if(debug) fprintf(stderr, "Matches : %lld\n", h_counter);
    for(long long i = h_old; i < h_counter; i++){
        if(recursive){
            printf("%s:",fname);
        }
        for(long long j = max(1ll,h_list[i].start); j < h_list[i].end; j++){
            printf("%c",data[j]);
        }
        printf("\n");
        //printf(" %lld %lld %lld\n",h_list[i].start,h_list[i].end,i);
    }
    h_old = h_counter;
    free(h_list);
    start = timenow(start,"printing");
    munmap(data, sb.st_size);
    close(fd);
}

/**
 * Recursively processes files in directory
 * @param path - Starting directory path
 */
void iterFiles(const string& path) {
    try {
        for(const auto& entry : fs::recursive_directory_iterator(path)) {
            if(fs::is_regular_file(entry)) {
                fileMap(entry.path());
            }
        }
    } catch(const fs::filesystem_error& e) {
        cerr << "Filesystem error: " << e.what() << '\n';
    }
}

__host__ void initializeGPUMemory() {
    auto start = chrono::high_resolution_clock::now();
    // First CUDA call - will initialize GPU
    char* d_pattern;
    Match *mlist;
    unsigned long long *d_counter;
    hipMalloc(&d_counter,sizeof(unsigned long long));
    // Add counter initialization
    unsigned long long init_val = 0;
    hipMemcpy(d_counter, &init_val, sizeof(unsigned long long), hipMemcpyHostToDevice);
    
    hipMalloc(&mlist,sizeof(Match)*MAX_MATCHES);
    hipMalloc(&d_pattern, pattern.size() + 1);
    hipMemcpy(d_pattern, pattern.c_str(), pattern.size() + 1, hipMemcpyHostToDevice);
    GpuConfig hostConfig = {
        d_pattern,
        patternSize,
        complement,
        mlist,
        d_counter,
        case_insensitive
    };
    hipMemcpyToSymbol(HIP_SYMBOL(c_config), &hostConfig, sizeof(GpuConfig));

}

string pattern_init(string pattern) {
    if(pattern[0] == '^') {
        match_start = true;
        pattern = pattern.substr(1);
    }
    else if(pattern[pattern.size()-1] == '$') {
        match_end = true;
        pattern = pattern.substr(0, pattern.size()-1);
    }
    // Initialize pattern for case insensitive search
    if(case_insensitive) {
        for(auto& c : pattern) {
            c = tolower(c);
        }
    }
    return pattern;
}

__host__ void cleanupGPUMemory() {
    // Get config from constant memory
    GpuConfig hostConfig;
    hipMemcpyFromSymbol(&hostConfig, HIP_SYMBOL(c_config), sizeof(GpuConfig));
    
    // Free pattern memory
    if(hostConfig.d_pattern) {
        hipFree(hostConfig.d_pattern);
    }
}

int main_handler(int argc, char *argv[]){
    auto startt = chrono::high_resolution_clock::now();
    fastio;
    startt = timenow(startt,"GPU Init");

    // Parse command line arguments
    if(argc < 2) {
        cerr << "Usage: " << argv[0] << " [-rv] pattern [file...]\n";
        return 1;
    }

    // Parse options
    int argIndex = 1;
    if(argv[argIndex][0] == '-') {
        string args = argv[argIndex];
        for(size_t i = 1; i < args.length(); i++) {
            switch(args[i]) {
                case 'r': recursive = true; break;
                case 'v': complement = true; break;
                case 'i': case_insensitive = true; break;
                default:
                    cerr << "Unknown option: -" << args[i] << '\n';
                    return 1;
            }
        }
        argIndex++;
    }

    // Initialize pattern
    pattern = pattern_init(argv[argIndex++]);
    patternSize = pattern.size();
    initializeGPUMemory();
    h_old = 0;

    // Process input (stdin or files)
    if(argc == argIndex) {
        searchstream(cin);
    } else {
        for(int i = argIndex; i < argc; i++) {
            recursive ? iterFiles(argv[i]) : fileMap(argv[i]);
        }
    }
    cleanupGPUMemory();
    auto finalt = chrono::high_resolution_clock::now();
    fprintf(stderr, "TOTAL_TIME:%ld us\n", chrono::duration_cast<chrono::microseconds>(finalt - startt).count());
    return 0;
}

int main(int argc, char* argv[]){
    hipFree(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Get properties of GPU 0
    MAX_THREADS = INT_MAX;
    if(argc == 1){ // Handle input stream
      string input;
      getline(cin, input);
      stringstream ss(input);
      string token;
      vector<string> tokens;
      while (ss >> token) {
        tokens.push_back(token);
      }
      char** new_argv = new char*[tokens.size() + 1];
      new_argv[0] = argv[0];
      for (size_t i = 0; i < tokens.size(); i++) {
        new_argv[i + 1] = const_cast<char*>(tokens[i].c_str());
      }
      argc = tokens.size() + 1;
      argv = new_argv;
      main_handler(argc,argv);
    }
    else main_handler(argc,argv); // Handle files
  }